#include "cuda_dll/src/memory_operations.cuh"
#include <hip/hip_runtime.h>

MemoryOperationsCode CUDAMalloc(void** out_p, size_t size)
{
    return (MemoryOperationsCode)hipMalloc(out_p, size);
}

MemoryOperationsCode CUDAFree (void* p)
{
    return (MemoryOperationsCode)hipFree(p);
}

MemoryOperationsCode TransmitToCUDA (void* host_p, void* cuda_p, size_t size)
{
    return (MemoryOperationsCode)hipMemcpy(cuda_p, host_p, size, hipMemcpyHostToDevice);
}

MemoryOperationsCode TransmitFromCUDA (void* host_p, void* cuda_p, size_t size)
{
    return (MemoryOperationsCode)hipMemcpy(host_p, cuda_p, size, hipMemcpyDeviceToHost);
}