#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_dll/src/build_world.cuh"
#include "cuda_dll/src/utils/cuda_thread_config.cuh"
#include "cuda_dll/src/utils/cuda_error_check.cuh"

// implementations in device code
#include "renderer/world/world3d.impl.cuh"


using namespace Kamanri::Utils;
using namespace Kamanri::Maths;
using namespace Kamanri::Renderer;
using namespace Kamanri::Renderer::World;
using namespace Kamanri::Renderer::World::__;

namespace __BuildWorld
{
	constexpr const char* LOG_NAME = STR(BuildWorld);

} // namespace BuildWorld$

__global__ void BuildPixelEntry(Kamanri::Renderer::World::World3D* p_world, unsigned int width, unsigned int height)
{
	size_t x = thread_index / height;
	size_t y = thread_index - x * height;

	if (x >= width || y >= height) return;
	
	p_world->__BuildForPixel((size_t)x, (size_t)y);
}

BuildWorldCode BuildWorld(Kamanri::Renderer::World::World3D* p_world, unsigned int width, unsigned int height)
{
	BuildPixelEntry
		thread_num(width * height)
		(p_world, width, height);
	
	auto res = hipDeviceSynchronize();

	CUDA_ERROR_CHECK(res, __BuildWorld::LOG_NAME);
	return BuildWorld$::CODE_NORM;
}


